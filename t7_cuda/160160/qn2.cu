
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>


#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))


__global__ void find_xor(int *input,int N, int* answer){
    int g_inx = blockIdx.x * blockDim.x + threadIdx.x;
    int b_inx = threadIdx.x;
    for(int i=blockDim.x/2;i>0;i=i/2){
        if((b_inx < i)&& (g_inx + i < N)){
            input[g_inx] = input[g_inx]^input[g_inx + i];
        }
        __syncthreads();
    }
    if(b_inx==0){
        atomicXor(answer,input[g_inx]);
    }

}

int main(int argc, char **argv){
    struct timeval start, end,s_start,s_end;
    int *h_input,*h_ans,*d_input,*d_ans;
    int N = atoi(argv[1]);
    srand(atoi(argv[2]));
    h_input = (int *)malloc(N * sizeof(int));
    h_ans = (int *)malloc(sizeof(int));
    h_ans[0] = 0;
    hipMalloc(&d_input, N*sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMalloc(&d_ans, sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    for(int i=0; i<N; i++){
        h_input[i] = (int)rand();
    }
    hipMemcpy(d_input,h_input, N*sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    hipMemcpy(d_ans,h_ans,sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    gettimeofday(&start, NULL);
    int blocks = N /1024;
    if(N % 1024)
        ++blocks;

    find_xor<<<blocks, 1024>>>(d_input,N,d_ans);
    CUDA_ERROR_EXIT("kernel invocation");
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);

    /* Copy back result*/
    hipMemcpy(h_ans,d_ans,sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");

    //printf("gpu Processsing =%ld microsecs\n", TDIFF(start, end));
    hipFree(d_input);
    hipFree(d_ans);

    gettimeofday(&s_start, NULL);
    int h_ans_seq=0;
    for (int i=0;i<N;i++){
         h_ans_seq = h_ans_seq^h_input[i];
    }
    gettimeofday(&s_end, NULL);

    printf("cpu Processsing =%ld microsecs\n", TDIFF(s_start, s_end));
    if(*h_ans==h_ans_seq){
         printf("congrats :) , Answer is %d\n",h_ans_seq);
     }
     else{
         printf(":( wrong answer, real: %d, yours: %d\n",h_ans_seq,*h_ans);
    }
    printf("%d\n",h_ans[0]);
    free(h_ans);
    free(h_input);

}

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void xor_it(int* darray, int* danswer, int num){
    int actual_id, tid;

    tid = threadIdx.x;
    actual_id = ( blockIdx.x * blockDim.x ) + tid;

    int i;
    for(i=(blockDim.x/2); i>0; i/=2){
        if(tid<i && actual_id+i<num){
            darray[actual_id] = darray[actual_id]^darray[actual_id+i];
        }
        __syncthreads();
    }
    if(tid==0){
        atomicXor(danswer, darray[actual_id]);
    }

}

int main(int argc, char **argv){

    int *harray, *darray, blocks;

    int *hanswer, *danswer;

    int num = atoi(argv[1]);
    int seed = atoi(argv[2]);

    harray = (int*) malloc(num*sizeof(int));

    hanswer = (int*) malloc(sizeof(int));
    *hanswer = 0;

    int i;
    srand(seed);
    for(i=0; i<num; i++){
        harray[i] = (int) rand();
    }

    hipMalloc(&darray, num*sizeof(int));
    hipMemcpy(darray, harray, num*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&danswer, sizeof(int));
    hipMemcpy(danswer, hanswer, sizeof(int), hipMemcpyHostToDevice);

    blocks = num/1024;

    if(num % 1024){
        blocks++;
    }

    xor_it<<<blocks, 1024>>>(darray, danswer, num);

    hipDeviceSynchronize();

    hipMemcpy(hanswer, danswer, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n",*hanswer);

    free(hanswer);
    free(harray);
    hipFree(darray);
    hipFree(danswer);

}
